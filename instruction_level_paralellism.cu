#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name			: Fundamentals of GPU programming
 Author			: David Celny
 Date			: 26.10.2018
 Description	: ILP behaviour validation
 Tasks			: demonstrate that the Instruction-level-parallelism (ILP)
can indeed increase the kernel performance (in terms of the time per arithmetic instruction)
				: Starting from having fewer number of threads per block
and gradually increasing it, show that the ILP becomes less efficient.
 ============================================================================
 */

#include <stddef.h>
#include <stdio.h>
#include "hip/hip_runtime_api.h"

#ifndef RND_WIDTH
#define RND_WIDTH 10
#endif // RND_WIDTH

#ifndef RND_SHIFT
#define RND_SHIFT 5
#endif // RND_SHIFT

#define CUDA_ERROR_CHECK
#define ONE_D_CASE

#ifndef ILP
#define ILP 1024   // number of the work ech thread calculate
#endif // ILP

/* === Error checking utility section === */
#define cudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define cudaSafeKernell()    __cudaCheckError( __FILE__, __LINE__ )

// constant memory declaration
__constant__ unsigned int d_size_x;
#ifdef ONE_D_CASE
__constant__ unsigned int d_size_y;
#endif

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "CUDA ERROR %i at %s: %i of type: %s\n",
                 err, file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "CUDA ERROR %i at %s: %i of type: %s\n",
        		err, file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "CUDA ERROR %i with sync at %s: %i of type: %s\n",
                 err, file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

/* === The program section === */
/* == the host code == */
__host__ __device__ inline void element_mulladd(unsigned int ind,float *matA, float *matB, float *matC)
/*
 * the device and host function for elementwise matrix multiplication+addition into matrixes
 */
{
	matC[ind] = matC[ind]*matA[ind]+ matB[ind]; // the very addition
}
#ifdef ONE_D_CASE
void matrix_mulladd_c (unsigned int size_x, float *matA, float *matB, float *matC)
/*
 * the CPU implementation of matrix multiplication+addition
 * the matrixes are in flattened form -> index as vectors
 *   								  -> beware row/coll order
 */
{
	unsigned int j;

	for (j = 0; j < size_x; j++)
	{ // walk through rows
		element_mulladd(j, matA,matB,matC);
	}
}
#else
void matrix_mulladd_c (unsigned int size_x, unsigned int size_y, float *matA, float *matB, float *matC)
/*
 * the CPU implementation of matrix multiplication+addition
 * the matrixes are in flattened form -> index as vectors
 *   								  -> beware row/coll order
 */
{
	unsigned int i,j;

	for (i = 0; i < size_y; i++)
	{ // operate on single row
		for (j = 0; j < size_x; j++)
		{ // walk through rows
			element_add(j + i*size_x, matA,matB,matC);
		}
	}
}
#endif
/* = the handling code = */
#ifdef ONE_D_CASE
void get_rnd_mat(unsigned int size_x, float *mat)
/*
 * fill the given array with random numbers
 *  random generator spread is set with RAND_WIDTH and RAND_SHIFT
 */
{
	unsigned int j;

	for (j = 0; j < size_x; j++)
	{ // walk through rows
		mat[j] = RND_WIDTH*(rand()/(float)(RAND_MAX)) - RND_SHIFT ;
	}
}
#else
void get_rnd_mat(unsigned int size_x, unsigned int size_y, float *mat)
/*
 * fill the given array with random numbers
 *  random generator spread is set with RAND_WIDTH and RAND_SHIFT
 */
{
	unsigned int i,j;

	for (i = 0; i < size_y; i++)
	{ // operate on single row
		for (j = 0; j < size_x; j++)
		{ // walk through rows
			mat[j + i*size_x] = RND_WIDTH*(rand()/(float)(RAND_MAX)) - RND_SHIFT ;
		}
	}
	return;
}
#endif
#ifdef ONE_D_CASE
void display_matrix(unsigned int size_x, float *mat)
/*
 * utility for displaying small matrixes
 */
{
	unsigned int j;

	printf("( ");
	for (j = 0; j < size_x; j++)
	{ // walk through rows
		printf("%f ",mat[j]);
	}
	printf(")\n");
}
#else
void display_matrix(unsigned int size_x, unsigned int size_y, float *mat)
/*
 * utility for displaying small matrixes
 */
{
	unsigned int i,j;
	for (i = 0; i < size_y; i++)
	{ // operate on single row
		printf("( ");
		for (j = 0; j < size_x; j++)
		{ // walk through rows
			printf("%f ",mat[j + i*size_x]);
		}
		printf(")\n");
	}
	printf("\n");

	return;
}
#endif
#ifdef ONE_D_CASE
int check_result(unsigned int size_x, float *mat_host, float*mat_dev, bool output=false)
/*
 * the verification function for result checking
 * 	samples both matrixes no mather how big
 * 			-> the problematic corner values are deffinitely sampled
 * 			-> the middle part is partially sampled (proportionaly to size)
 * 	! beware oversampling for smaller matrixes
 * 	default output is used for printing
 */
{
	unsigned int i, tmp;
	const unsigned int idx_cnt = 3+log2(1.0*size_x); //sampling quantity
	unsigned int sample_idx[idx_cnt]; // the sampled indexes

	// corner indexes
	sample_idx[0] = 0;
	sample_idx[1] = size_x - 1;
	// middle siede indexes
	sample_idx[2] = size_x/2;
	// the sampled middle of matrix
	if (size_x>2)
	{
		for (i = 3; i < idx_cnt; i++)
		{
			tmp = (int)((size_x)*(rand()/(float)(RAND_MAX)));			// random x index of size_x
			sample_idx[i] = tmp;
		}
	}
// 	printf(" *** Sampling %d elements from matrix: ***\n     2 corners, 1 middle, %d insides\n", idx_cnt, idx_cnt-3);
	for (i = 0; i < idx_cnt; i++)
	{
		tmp = mat_dev[sample_idx[i]] - mat_host[sample_idx[i]]; // perform subtraction
		if (tmp!= 0) // beware the epsilon differences - not solved
		{// SELFNOTE for (-5,5) the situation seems fine
			printf("Difference in CPU/GPU comparison at index: %d \n",sample_idx[i]);
			return -1;
		}
		else if (output)
		{
			printf("at [x=%4d] D=%16.12f H=%16.12f \n",sample_idx[i],mat_dev[sample_idx[i]], mat_host[sample_idx[i]]);
		}
	}
	return 0;
}
#else
int check_result(unsigned int size_x, unsigned int size_y, float *mat_host, float*mat_dev, bool output=false)
/*
 * the verification function for result checking
 * 	samples both matrixes no mather how big
 * 			-> the problematic corner values are deffinitely sampled
 * 			-> the middle part is partially sampled (proportionaly to size)
 * 	! beware oversampling for smaller matrixes
 * 	default output is used for printing
 */
{
	unsigned int i, tmp;
	const unsigned int idx_cnt = 8+log2(1.0*size_x*size_y); //sampling quantity
	unsigned int sample_idx[idx_cnt]; // the sampled indexes

	// corner indexes
	sample_idx[0] = 0;
	sample_idx[1] = size_x - 1;
	sample_idx[2] = (size_y - 1)*size_x;
	sample_idx[3] = size_x*size_y - 1;
	// middle siede indexes
	sample_idx[4] = size_x/2;
	sample_idx[5] = size_y/2 *size_x;
	sample_idx[6] = size_y/2*(size_x+1);
	sample_idx[7] = size_x*size_y - size_x/2;
	// the sampled middle of matrix
	if (size_x>2 && size_y>2)
	{
		for (i = 8; i < idx_cnt; i++)
		{
			tmp = (int)((size_x)*(rand()/(float)(RAND_MAX)));			// random x index of size_x
			tmp += (int)((size_y)*(rand()/(float)(RAND_MAX)))*size_x; // and random y index of size_y
			sample_idx[i] = tmp;
		}
	}
	printf(" *** Sampling %d elements from matrix: ***\n     4 corners, 4 middle sides, %d insides\n", idx_cnt, idx_cnt-8);
	for (i = 0; i < idx_cnt; i++)
	{
		tmp = mat_dev[sample_idx[i]] - mat_host[sample_idx[i]]; // perform subtraction
		if (tmp!= 0) // beware the epsilon differences - not solved
		{// SELFNOTE for (-5,5) the situation seems fine
			printf("Difference in CPU/GPU comparison at index: %d, x=%d y=%d \n",sample_idx[i],sample_idx[i]/size_x,sample_idx[i]%size_x );
			return -1;
		}
		else if (output)
		{
			printf("at [x=%4d, y=%4d] D=%16.12f H=%16.12f \n",sample_idx[i]/size_x,sample_idx[i]%size_x,mat_dev[sample_idx[i]], mat_host[sample_idx[i]]);
		}
	}
	return 0;
}
#endif
/* == the device code == */
__global__ void matrix_mulladd_g (float *matA, float *matB, float *matC)
/*
 * The kernel for multiplication+addition of matrix
 * ! elementwise addition with size input size_x, size_y
 * 	 expected to be called as two dimensional grid
 * !! the ifdef generalized also to single dimension
 *   the matrixes are in flattened form -> indexed as vectors
 *   									-> beware row/coll order
 */
{
	unsigned int idx =  ILP*(blockDim.x * blockIdx.x + threadIdx.x); // x_dim index
	if(idx> d_size_x) return;
#ifndef ONE_D_CASE
	idx += (blockDim.y * blockIdx.y + threadIdx.y)*d_size_x; // y_dim index increment
	if(idx> d_size_x*d_size_y) return;
#endif
#pragma unroll
	for(int i=0; i<ILP; i++)
	{
		element_mulladd(idx+i, matA,matB,matC);
	}

}

int main( int argc, char *argv[] )
/*
 * main function executing the kernell call and GPU properties output
 */
{
	const long int seed = 123456789;

	const unsigned int size_x = 4096;

// 	unsigned int thread_count_x = 32; // number of threads used per block in x - valid for Block ILP only

#ifdef ONE_D_CASE
	size_t size_n = size_x*sizeof(float);
#else
	const unsigned int size_y = 100;
	size_t size_n = size_x*size_y*sizeof(float);
	unsigned int thread_count_y = 32; // number of threads used per block in y
#endif

	int awailable_device; // the device number ! expect only single GPU coprocessor presence to work as intended

	// host variables
	float *matA = NULL;
	float *matB = NULL;
	float *matC = NULL;
	float *matC_dev;
	// device variables
	float *d_matA;
	float *d_matB;
	float *d_matC;

	// timing
	clock_t cpu_start, cpu_stop;
	hipEvent_t gpu_start, gpu_stop;
	float cpu_time, gpu_time;

	/* get and display device infromation section */
	hipDeviceProp_t device_prop;
	cudaSafeCall( hipGetDevice(&awailable_device)); //get the device count
	cudaSafeCall( hipGetDeviceProperties(&device_prop, awailable_device)); // get the last device properties

// 	printf("*** Coprocessor %s information ***\n", device_prop.name);
// 	printf("*** SM: %i, Gmem: %d MB, Smem/B: %d kB, Cmem: %d kB ***\n",(int)device_prop.multiProcessorCount
// 																	  ,(int)device_prop.totalGlobalMem/1024/1024
// 																	  ,(int)device_prop.sharedMemPerBlock/1024
// 																	  ,(int)device_prop.totalConstMem/1024);
	/* initialization section */
	srand(seed);
	matA = (float*) malloc(size_n);
	matB = (float*) malloc(size_n);
	matC = (float*) malloc(size_n);
	matC_dev = (float*) malloc(size_n);
#ifdef ONE_D_CASE
	get_rnd_mat(size_x,matA);
// 	display_matrix(size_x, matA);
	get_rnd_mat(size_x,matB);
// 	display_matrix(size_x, matB);
	get_rnd_mat(size_x,matC);
// 	display_matrix(size_x, matC);
#else
	get_rnd_mat(size_x,size_y,matA);
//	display_matrix(size_x, size_y, matA);
	get_rnd_mat(size_x,size_y,matB);
//	display_matrix(size_x, size_y, matB);
	get_rnd_mat(size_x,size_y,matC);
//	display_matrix(size_x, size_y, matC);
#endif
	// the device init
	cudaSafeCall(hipMalloc((void**)&d_matA, size_n));
	cudaSafeCall(hipMalloc((void**)&d_matB, size_n));
	cudaSafeCall(hipMalloc((void**)&d_matC, size_n));
	// constant memory move
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_size_x), &size_x, sizeof(unsigned int)));
#ifndef ONE_D_CASE
	cudaSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(d_size_y), &size_y, sizeof(unsigned int)));
#endif
	// the data move host->device
	cudaSafeCall(hipMemcpy(d_matA, matA, size_n, hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_matB, matB, size_n, hipMemcpyHostToDevice));
	cudaSafeCall(hipMemcpy(d_matC, matC, size_n, hipMemcpyHostToDevice));
	// timing
	cudaSafeCall(hipEventCreate(&gpu_start));
	cudaSafeCall(hipEventCreate(&gpu_stop));

	/* Kernell call section */
#ifdef ONE_D_CASE
	/* for block case ILP */
// 	dim3 thread_dim(thread_count_x); 
// 	dim3 block_dim(ceil((float)(size_x/ILP)/thread_count_x)); 
	/* for thread case ILP */
	dim3 thread_dim(ILP); 
	dim3 block_dim(ceil((float)size_x/ILP)); 
#else
	/* for block case ILP */
// 	dim3 thread_dim(thread_count_x,thread_count_y);
// 	dim3 block_dim(ceil((float)(size_x/ILP)/thread_count_x),ceil((float)size_y/thread_count_y));
	/* for thread case ILP */
	dim3 thread_dim(ILP,thread_count_y);
	dim3 block_dim(ceil((float)size_x/ILP),ceil((float)size_y/thread_count_y));
#endif
	cudaSafeCall(hipEventRecord(gpu_start, 0));

	matrix_mulladd_g<<<block_dim,thread_dim>>>(d_matA, d_matB, d_matC);
	hipDeviceSynchronize();
	cudaSafeCall(hipEventRecord(gpu_stop, 0));
	cudaSafeCall(hipEventSynchronize(gpu_stop));
//	cudaSafeKernell(); // not to be included in speed measurement

	cudaSafeCall(hipEventElapsedTime(&gpu_time,gpu_start,gpu_stop));
	// the data move device->host
	cudaSafeCall(hipMemcpy(matC_dev, d_matC, size_n, hipMemcpyDeviceToHost));

	// event cleaup
	cudaSafeCall(hipEventDestroy(gpu_start));
	cudaSafeCall(hipEventDestroy(gpu_stop));
	/* CPU section */
	cpu_start = clock();
#ifdef ONE_D_CASE
	matrix_mulladd_c(size_x, matA, matB, matC);
#else
	matrix_mulladd_c(size_x, size_y, matA, matB, matC);
#endif
	cpu_stop = clock();

	cpu_time = (cpu_stop-cpu_start)/((float)CLOCKS_PER_SEC);
	/* execution statistics section */
	printf("*** level of parallelization ***\n");
#ifdef ONE_D_CASE
	printf("*** vector: %d ***\n", size_x);
	printf("*** block: %d ***\n", block_dim.x);
	printf("*** thread: %d ***\n", thread_dim.x);
#else
	printf("*** matrix: %d,%d ***\n", size_x, size_y);
	printf("*** block: %d,%d ***\n", block_dim.x, block_dim.y);
	printf("*** thread: %d,%d ***\n", thread_dim.x, thread_dim.y);
#endif
	printf("*** element per thread: %d ***\n", ILP);
	printf("*** time measurement ***\n");
	printf("*** CPU: %f ms\n",cpu_time*1000);
	printf("*** GPU: %f ms\n",gpu_time);
	printf("*** speedup: %f \n",cpu_time*1000/gpu_time);

	// print for batch execution
	// printf("%16.12f , %16.12f, %16.12f \n",cpu_time*1000,gpu_time, cpu_time*1000/gpu_time);
	
	/* controll section */
#ifdef ONE_D_CASE
// 	display_matrix(size_x, matC);
// 	display_matrix(size_x, matC_dev);
	int result= check_result(size_x, matC, matC_dev, false);
#else

//	display_matrix(size_x, size_y, matC);
//	display_matrix(size_x, size_y, matC_dev);
	int result= check_result(size_x, size_y, matC, matC_dev, false);
#endif

	/* celanup section */
	free(matA);
	free(matB);
	free(matC);
	free(matC_dev);
	hipFree(d_matA);
	hipFree(d_matB);
	hipFree(d_matC);

	return result;
}
